#include "hip/hip_runtime.h"
#include"gpuVector.h"
//#include"hipblas.h"
#include"hip/hip_runtime.h"
//#include"lib.cuh"
#include"vector"
#include"gpuVector.cuh"

//#define __DEBUG_GVECTOR

using namespace gv;

void test_gVector(void) {
	gVector<float>::Init(100000);
	{
		gVector<float> v1(10000), v2(10000), v3(10000);
		v1.set(1);
		v2.set(1.5);
		v3.set(3);
		//constexpr bool val = decltype(v1 - v2)::is_exp;
		gVector<float> v4 = (v1 - v2) / v3 + v1 * v1 - 3;

		std::cout << "max of v4 " << v4.max() << std::endl;
		std::cout << "min of v4 " << v4.min() << std::endl;
		std::cout << "sum of v4 " << v4.sum() << std::endl;
		std::cout << "norm of v4 " << v4.norm() << std::endl;
		std::cout << "norm of v4 / 2 " << (v4 / 2.f).norm() << std::endl;


		// test dot
		std::cout << "v2'*(v1+v3) = " << v2.dot(v1 + v3) << std::endl;

		// test map
		gVector<float> v5 = gVector<float>::Map(v4.data(), 1000);
		std::cout << "max of v5 " << v5.max() << std::endl;
		std::cout << "min of v5 " << v5.min() << std::endl;
		std::cout << "norm of v5 " << v5.norm() << std::endl;
		std::cout << "norm of v5 * 2 " << (v5 * 2.f).norm() << std::endl;

		// test index access
		gVector<float> v6 = gVector<float>::Map(v5.data(), 10);
		std::cout << "old value : " << std::endl;
		for (int i = 0; i < 10; i++) {
			Scalar val = v6[i];
			printf("v6[%d] = %f\n", i, val);
			v6[i] = val - 1;
		}

		std::cout << "new value : " << std::endl;
		for (int i = 0; i < 10; i++) {
			Scalar val = v6[i];
			printf("v6[%d] = %f\n", i, val);
		}

		// test max/minimize 
		v5.maximize(1);
		printf("maximized v5 max = %f\n", v5.max());
		printf("maximized v5 min = %f\n", v5.min());

		v5.minimize(-1);
		printf("minimized v5 max = %f\n", v5.max());
		printf("minimized v5 min = %f\n", v5.min());

		printf("max(v5,2) norm = %f\n", v5.max(2).norm());

		// test concate
		gVector<float> v7 = v1.concated(v2, v3);
		printf("size of v7 = %d\n", v7.size());
		printf("max of v7 = %f \n", v7.max());
		printf("min of v7 = %f \n", v7.min());

		gVector<float> v8;
		v8.concate(gVector<float>(2, 1), gVector<float>(2, 2), gVector<float>(2, 3), 4, 4);
		printf("v8 = \n");
		for (int i = 0; i < v8.size(); i++) {
			printf("%f\n", Scalar(v8[i]));
		}
	}
}




